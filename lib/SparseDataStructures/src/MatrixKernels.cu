
#include <hip/hip_runtime.h>
__global__ void matrixVectorMultKernel(const float* matrix, const float* v1, float* rv, const unsigned rows, const unsigned cols)
{
    const unsigned row = blockIdx.x * blockDim.x + threadIdx.x;

    if(row >= rows) return;

    float acc = 0;

    for(unsigned i = 0u; i < cols; i++ )
    {
        acc+= matrix[row * cols + i] * v1[i];
    }    

    rv[row] = acc;
}