#include "hip/hip_runtime.h"
#include "CommitOriginalDataStructure.hpp"

#include <iomanip>

/* ========================================================================================== */
template<typename T>
bool areNearlyEqual(T a, T b) {
    const T normal_min = std::numeric_limits<T>::min();
    const T relative_error = 0.000009;
    if (!std::isfinite(a) || !std::isfinite(b))
    {
        return false;
    }

    T diff = std::abs(a - b);
    if (diff <= normal_min) 
        return true;

    T abs_a = std::abs(a);
    T abs_b = std::abs(b);

    return (diff / std::max(abs_a, abs_b)) <= relative_error;
}

template<typename T>
bool verifyCorrectness(const std::vector<T>& correct,const std::vector<T>& obtained)
{
    if (correct.size() != obtained.size())
    {
        std::cout << "Correct and obtained vectors don't have same size!" << std::endl;
        return false;
    }

    for(size_t i = 0;i < correct.size();i++)
    {
        if(!areNearlyEqual<T>(correct[i],obtained[i]))
        {
            std::cout << std::fixed << std::setprecision(6) <<
                      "Error found : correct[" << i << "] = " << correct[i] << ", obtained[" << i << "] = " << obtained[i] << std::endl; 
            return false;
        }
    }
    return true;
}

void printResult(const std::string& message,bool correctness,float kernelTime,float totalTime){

    const std::string upperSepSx  = "------------------ ";
    const std::string upperSepDx  = " ------------------";
    const std::string downerSep(upperSepSx.length()*2+message.length(),'-'); 

    std::cout << upperSepSx << message << upperSepDx                       << std::endl
              << "| correct        => " << ((correctness)? "true":"false") << std::endl
              << "| kernel time    => " << kernelTime << " ms"             << std::endl
              << "| total  time    => " << totalTime  << " ms"             << std::endl
              << downerSep                                                 << std::endl;  

}
/* ========================================================================================== */

#define CUDAERRCHECK(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

__global__ void commitMatrixMultiplication(
    const uint32_t* icfDevice, const uint32_t* icvDevice, const uint16_t* icoDevice, const float* iclDevice, int nR,
    const uint32_t* ecvDevice, const uint16_t* ecoDevice, int nT,
    const uint32_t* isovDevice, int nI,
    const float* wmrSFPDevice, const float* wmhSFPDevice, const float* isoSFPDevice,int ndirs,
    const int* icIndexesDevice, const int* ecIndexesDevice, const int* isoIndexesDevice,
    float* xDevice,
    float* yDevice
)
{
    const int voxel       = blockIdx.x * blockDim.x;
    const int voxelOffset = voxel + threadIdx.x;

    /* IC */
    __shared__ float acc[100];
    acc[ threadIdx.x ] = 0.0f;

    for(int icsegment = voxel==0?0:icIndexesDevice[voxel-1]; icsegment < icIndexesDevice[voxel]; icsegment++)
    {
        for(int radii = 0; radii < nR; radii++)
        {
            acc[threadIdx.x] += 
                iclDevice[icsegment] * 
                wmrSFPDevice[(radii*ndirs * blockDim.x) + (icoDevice[icsegment]* blockDim.x + threadIdx.x)] * 
                xDevice[icfDevice[icsegment] + radii];
        }
    }
    yDevice[voxelOffset] += acc[threadIdx.x]; // TODO: IDEA, COMPUTAZIONI NEL CICLO IN SHARED MEMORY, 1 SOLA SCRITTURA IN GLOBAL DOPO
    /* EC */
    for(int ecsegment = voxel==0?0:ecIndexesDevice[voxel-1]; ecsegment < ecIndexesDevice[voxel]; ecsegment++)
    {
        for(int tortuosity = 0; tortuosity < nT; tortuosity++)
        {
        }
    }
    /* ISO */ 
    for(int isosegment = voxel==0?0:isoIndexesDevice[voxel-1]; isosegment < isoIndexesDevice[voxel]; isosegment++)
    {
        for(int iso = 0; iso < nI; iso++)
        {
        }
    }
}

void CommitOriginalDataStructure::gpuMatrixMultiplication()
{
    hipEvent_t totalStart,totalStop;
    hipEventCreate(&totalStart);
    hipEventCreate(&totalStop);

    hipEvent_t kernelStart,kernelStop;
    hipEventCreate(&kernelStart);
    hipEventCreate(&kernelStop);

    hipEventRecord(totalStart);

    /* IC */
    uint32_t* icfDevice; uint32_t* icvDevice; uint16_t* icoDevice; float* iclDevice;

    CUDAERRCHECK(hipMalloc(&icfDevice,sizeof(uint32_t)*icf.size()))
    CUDAERRCHECK(hipMalloc(&icvDevice,sizeof(uint32_t)*icv.size()))
    CUDAERRCHECK(hipMalloc(&icoDevice,sizeof(uint16_t)*ico.size()))
    CUDAERRCHECK(hipMalloc(&iclDevice,sizeof(float)*icl.size()))

    CUDAERRCHECK(hipMemcpy(icfDevice,icf.data(),sizeof(uint32_t)*icf.size(),hipMemcpyHostToDevice))
    CUDAERRCHECK(hipMemcpy(icvDevice,icv.data(),sizeof(uint32_t)*icv.size(),hipMemcpyHostToDevice))
    CUDAERRCHECK(hipMemcpy(icoDevice,ico.data(),sizeof(uint16_t)*ico.size(),hipMemcpyHostToDevice))
    CUDAERRCHECK(hipMemcpy(iclDevice,icl.data(),sizeof(float)*icl.size(),hipMemcpyHostToDevice))

    /* EC */
    uint32_t* ecvDevice; uint16_t* ecoDevice;

    CUDAERRCHECK(hipMalloc(&ecvDevice,sizeof(uint32_t)*ecv.size()))
    CUDAERRCHECK(hipMalloc(&ecoDevice,sizeof(uint16_t)*eco.size()))

    CUDAERRCHECK(hipMemcpy(ecvDevice,ecv.data(),sizeof(uint32_t)*ecv.size(),hipMemcpyHostToDevice))
    CUDAERRCHECK(hipMemcpy(ecoDevice,eco.data(),sizeof(uint16_t)*eco.size(),hipMemcpyHostToDevice))

    /* ISO */
    uint32_t* isovDevice;

    CUDAERRCHECK(hipMalloc(&isovDevice,sizeof(uint32_t)*isov.size()))

    CUDAERRCHECK(hipMemcpy(isovDevice,isov.data(),sizeof(uint32_t)*isov.size(),hipMemcpyHostToDevice))

    /* LOOKUP TABLE */
    float* wmrSFPDevice;float* wmhSFPDevice;float* isoSFPDevice;

    CUDAERRCHECK(hipMalloc(&wmrSFPDevice,sizeof(float)*wmrSFP.size()))
    CUDAERRCHECK(hipMalloc(&wmhSFPDevice,sizeof(float)*wmhSFP.size()))
    CUDAERRCHECK(hipMalloc(&isoSFPDevice,sizeof(float)*isoSFP.size()))

    CUDAERRCHECK(hipMemcpy(wmrSFPDevice,wmrSFP.data(),sizeof(float)*wmrSFP.size(),hipMemcpyHostToDevice))
    CUDAERRCHECK(hipMemcpy(wmhSFPDevice,wmhSFP.data(),sizeof(float)*wmhSFP.size(),hipMemcpyHostToDevice))
    CUDAERRCHECK(hipMemcpy(isoSFPDevice,isoSFP.data(),sizeof(float)*isoSFP.size(),hipMemcpyHostToDevice))

    /* HELPER ARRAYS */
    int* icIndexesDevice;int* ecIndexesDevice;int* isoIndexesDevice;

    CUDAERRCHECK(hipMalloc(&icIndexesDevice,sizeof(int)*icIndexes.size()))
    CUDAERRCHECK(hipMalloc(&ecIndexesDevice,sizeof(int)*ecIndexes.size()))
    CUDAERRCHECK(hipMalloc(&isoIndexesDevice,sizeof(int)*isoIndexes.size()))

    CUDAERRCHECK(hipMemcpy(icIndexesDevice,icIndexes.data(),sizeof(int)*icIndexes.size(),hipMemcpyHostToDevice))
    CUDAERRCHECK(hipMemcpy(ecIndexesDevice,ecIndexes.data(),sizeof(int)*ecIndexes.size(),hipMemcpyHostToDevice))
    CUDAERRCHECK(hipMemcpy(isoIndexesDevice,isoIndexes.data(),sizeof(int)*isoIndexes.size(),hipMemcpyHostToDevice))

    /* INPUT */
    float* xDevice;
    
    CUDAERRCHECK(hipMalloc(&xDevice,sizeof(float)*input.size()))

    CUDAERRCHECK(hipMemcpy(xDevice,input.data(),sizeof(float)*input.size(),hipMemcpyHostToDevice))

    /* RESULT */
    float* yDevice;

    CUDAERRCHECK(hipMalloc(&yDevice,sizeof(float)*output.size()))
    
    CUDAERRCHECK(hipMemset(yDevice,0.0f,sizeof(float)*output.size()))

    /* BLOCKS AND THREAD ORGANIZATION */
    const int blocks = _nV;
    const int threadsPerBlock = _nS;

    dim3 dimGrid(blocks,1,1);
    dim3 dimBlock(threadsPerBlock,1,1);

    hipEventRecord(kernelStart);
    commitMatrixMultiplication<<<dimGrid,dimBlock>>>(
        icfDevice,icvDevice,icoDevice,iclDevice,_nR,
        ecvDevice,ecoDevice,_nT,
        isovDevice,_nI,
        wmrSFPDevice,wmhSFPDevice,isoSFPDevice,_ndirs,
        icIndexesDevice,ecIndexesDevice,isoIndexesDevice,
        xDevice,
        yDevice
    );
    hipEventRecord(kernelStop);

    /* COPYING BACK THE RESULT */
    std::vector<float> obtainedResult(output.size(),0.0f);
    CUDAERRCHECK(hipMemcpy(obtainedResult.data(),yDevice,sizeof(float)*output.size(),hipMemcpyDeviceToHost))

    /* FREEING MEMORY */
    hipFree(icfDevice);hipFree(icvDevice);hipFree(icoDevice);hipFree(iclDevice);
    hipFree(ecvDevice);hipFree(ecoDevice);
    hipFree(isovDevice);
    hipFree(wmrSFPDevice);hipFree(wmhSFPDevice);hipFree(isoSFPDevice);
    hipFree(icIndexesDevice);hipFree(ecIndexesDevice);hipFree(isoIndexesDevice);

    hipEventRecord(totalStop);

    /* VERIFYING CORRECTNESS OF THE RESULT */
    hipEventSynchronize(kernelStop);
    float kernelMilliseconds = 0;
    hipEventElapsedTime(&kernelMilliseconds,kernelStart,kernelStop);
    
    hipEventSynchronize(totalStop);
    float totalMilliseconds = 0;
    hipEventElapsedTime(&totalMilliseconds,totalStart,totalStop);
    
    printResult("Gpu matrix multiplication", verifyCorrectness<float>(output,obtainedResult),kernelMilliseconds,totalMilliseconds);

    hipDeviceReset();
}