#include "hip/hip_runtime.h"
#include "CommitOriginalDataStructure.hpp"

#include <iomanip>

/* ========================================================================================== */
template<typename T>
bool areNearlyEqualGpu(T a, T b) {
    const T normal_min = std::numeric_limits<T>::min();
    const T relative_error = 0.00001;
    if (!std::isfinite(a) || !std::isfinite(b))
    {
        return false;
    }

    T diff = std::abs(a - b);
    if (diff <= normal_min) 
        return true;

    T abs_a = std::abs(a);
    T abs_b = std::abs(b);

    return (diff / std::max(abs_a, abs_b)) <= relative_error;
}

template<typename T>
bool verifyCorrectnessGpu(const std::vector<T>& correct,const std::vector<T>& obtained)
{
    if (correct.size() != obtained.size())
    {
        std::cout << "Correct and obtained vectors don't have same size!" << std::endl;
        return false;
    }

    for(size_t i = 0;i < correct.size();i++)
    {
        if(!areNearlyEqualGpu<T>(correct[i],obtained[i]))
        {
            std::cout << std::fixed << std::setprecision(6) <<
                      "Error found : correct[" << i << "] = " << correct[i] << ", obtained[" << i << "] = " << obtained[i] << ", abs err : " << std::abs(correct[i] - obtained[i])<< std::endl; 
            return false;
        }
    }
    return true;
}

float gpuCalculateAverageAbsoluteError(const std::vector<float>& correct,const std::vector<float>& obtained)
{
    if (correct.size() != obtained.size())
    {
        std::cout << "Correct and obtained vectors don't have same size!" << std::endl;
        return false;
    }

    float accAbsErr = 0.0f;

    for(size_t i = 0;i < correct.size();i++)
    {
        accAbsErr += std::abs(correct[i] - obtained[i]);
    }
    return accAbsErr/static_cast<float>(correct.size());
}

void printResultGpu(const std::string& message,const std::vector<float>& correct,const std::vector<float>& obtained,bool correctness,float kernelTime,float totalTime){

    const std::string upperSepSx  = "------------------ ";
    const std::string upperSepDx  = " ------------------";
    const std::string downerSep(upperSepSx.length()*2+message.length(),'-'); 

    std::cout << upperSepSx << message << upperSepDx                                       << std::endl
              << "| correct        => " << ((correctness)? "true":"false")                    << std::endl
              << "| kernel time    => " << kernelTime << " ms"                             << std::endl
              << "| total  time    => " << totalTime  << " ms"                             << std::endl
              << "| avg abs err    => " << gpuCalculateAverageAbsoluteError(correct,obtained) << std::endl
              << downerSep                                                                 << std::endl;

}
/* ========================================================================================== */

#define CUDAERRCHECK(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

/*
    QUADDRO P6000 => 
                        max_threads_per_sm : 2048
                        max_blocks_per_sm  : 32
                        -------------------------
                        threads per block to achieve max occupancy : 64

                        n_sm : 30
                        -------------------------
                        to achieve full GPU occupancy : 960 (or a multiple of it) blocks of 64 threads

    RTX 2060      => 
                        max_threads_per_sm : 1024
                        max_blocks_per_sm  : 16
                        -------------------------
                        threads per block to achieve max occupancy : 64

                        n_sm : 30
                        -------------------------
                        to achieve full GPU occupancy : 480 (or a multiple of it) blocks of 64 threads 
*/

__global__ void commitMatrixMultiplication(
    uint32_t* icfDevice, float* iclDevice,
    int* icIndexesDevice,
    float* xDevice,
    float* yDevice
)
{
    /* SHARED MEMORY BUFFERS */
    extern __shared__ float buffer[];
    int* xBuffer = (int*)buffer;
    float* lengthsBuffer = &buffer[32];

    const int voxel = blockIdx.x;

    const int startIcSegment = (voxel == 0) ? 0 : icIndexesDevice[voxel-1];
    const int endIcSegment   = icIndexesDevice[voxel];
    const int totalIcSegments  = endIcSegment - startIcSegment;

    const int TOTAL_IC_TILES = 1 + ((totalIcSegments-1)/32);

    float result = 0.0f;

    for(int tile = 0; tile < TOTAL_IC_TILES; tile++)
    {
        int segmentIndex = startIcSegment + tile * 32 + threadIdx.x;


        /*==================================================================*/
        /*                 PART TO OPTIMIZE                                 */

        if(segmentIndex < endIcSegment)
        {
            xBuffer[threadIdx.x]  = xDevice[icfDevice[segmentIndex]];
            lengthsBuffer[threadIdx.x] = iclDevice[segmentIndex];
        }

        /* CALCULATING MULTIPLICATION */
        float accumulator = 0.0f;
        
        if(segmentIndex < endIcSegment)
        {
            accumulator = xBuffer[threadIdx.x] * lengthsBuffer[threadIdx.x];
        }
        /*==================================================================*/

        /* REDUCTION */
        for (int offset = 16; offset > 0; offset /= 2)
            accumulator += __shfl_down_sync(0xffffffff, accumulator, offset);
        
        result += accumulator;
    }

    /* WRITING OUT THE RESULT */
    if(threadIdx.x == 0)
    {
        yDevice[voxel] = result;
    }

}

void CommitOriginalDataStructure::gpuMatrixMultiplication()
{
    hipEvent_t totalStart,totalStop;
    hipEventCreate(&totalStart);
    hipEventCreate(&totalStop);

    hipEvent_t kernelStart,kernelStop;
    hipEventCreate(&kernelStart);
    hipEventCreate(&kernelStop);

    hipEventRecord(totalStart);

    /* IC */
    uint32_t* icfDevice; float* iclDevice;

    CUDAERRCHECK(hipMalloc(&icfDevice,sizeof(uint32_t)*icf.size()))
    CUDAERRCHECK(hipMalloc(&iclDevice,sizeof(float)*icl.size()))

    CUDAERRCHECK(hipMemcpy(icfDevice,icf.data(),sizeof(uint32_t)*icf.size(),hipMemcpyHostToDevice))
    CUDAERRCHECK(hipMemcpy(iclDevice,icl.data(),sizeof(float)*icl.size(),hipMemcpyHostToDevice))

    /* HELPER INDEXES */
    int* icIndexesDevice;

    CUDAERRCHECK(hipMalloc(&icIndexesDevice,sizeof(int)*icIndexes.size()))
    CUDAERRCHECK(hipMemcpy(icIndexesDevice,icIndexes.data(),sizeof(int)*icIndexes.size(),hipMemcpyHostToDevice))

    /* INPUT */
    float* xDevice;
    
    CUDAERRCHECK(hipMalloc(&xDevice,sizeof(float)*input.size()))

    CUDAERRCHECK(hipMemcpy(xDevice,input.data(),sizeof(float)*input.size(),hipMemcpyHostToDevice))

    /* RESULT */
    float* yDevice;

    CUDAERRCHECK(hipMalloc(&yDevice,sizeof(float)*output.size()))
    
    CUDAERRCHECK(hipMemset(yDevice,0.0f,sizeof(float)*output.size()))

    /* BLOCKS AND THREAD ORGANIZATION */
    const int blocks = _nV;
    const int threadsPerBlock = 32;

    dim3 dimGrid(blocks,1,1);
    dim3 dimBlock(threadsPerBlock,1,1);

    hipEventRecord(kernelStart);
    commitMatrixMultiplication<<<dimGrid,dimBlock,2*32*sizeof(float)>>>(
        icfDevice,iclDevice,
        icIndexesDevice,
        xDevice,
        yDevice
    );
    hipEventRecord(kernelStop);

    /* COPYING BACK THE RESULT */
    std::vector<float> obtainedResult(output.size(),0.0f);
    CUDAERRCHECK(hipMemcpy(obtainedResult.data(),yDevice,sizeof(float)*output.size(),hipMemcpyDeviceToHost))

    /* FREEING MEMORY */
    hipFree(icfDevice);hipFree(iclDevice);
    hipFree(icIndexesDevice);

    hipEventRecord(totalStop);

    /* VERIFYING CORRECTNESS OF THE RESULT */
    hipEventSynchronize(kernelStop);
    float kernelMilliseconds = 0;
    hipEventElapsedTime(&kernelMilliseconds,kernelStart,kernelStop);
    
    hipEventSynchronize(totalStop);
    float totalMilliseconds = 0;
    hipEventElapsedTime(&totalMilliseconds,totalStart,totalStop);
    
    printResultGpu("Gpu matrix multiplication",output,obtainedResult,verifyCorrectnessGpu<float>(output,obtainedResult),kernelMilliseconds,totalMilliseconds);

    hipDeviceReset();
}