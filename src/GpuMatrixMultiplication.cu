#include "hip/hip_runtime.h"
#include "CommitOriginalDataStructure.hpp"

#include <iomanip>
#include <cassert>

/* ========================================================================================== */
template<typename T>
bool areNearlyEqualGpu(T a, T b) {
    const T normal_min = std::numeric_limits<T>::min();
    const T relative_error = 0.00001;
    if (!std::isfinite(a) || !std::isfinite(b))
    {
        return false;
    }

    T diff = std::abs(a - b);
    if (diff <= normal_min) 
        return true;

    T abs_a = std::abs(a);
    T abs_b = std::abs(b);

    return (diff / std::max(abs_a, abs_b)) <= relative_error;
}

template<typename T>
bool verifyCorrectnessGpu(const std::vector<T>& correct,const std::vector<T>& obtained)
{
    if (correct.size() != obtained.size())
    {
        std::cout << "Correct and obtained vectors don't have same size!" << std::endl;
        return false;
    }

    for(size_t i = 0;i < correct.size();i++)
    {
        if(!areNearlyEqualGpu<T>(correct[i],obtained[i]))
        {
            std::cout << std::fixed << std::setprecision(6) <<
                      "Error found : correct[" << i << "] = " << correct[i] << ", obtained[" << i << "] = " << obtained[i] << ", abs err : " << std::abs(correct[i] - obtained[i])<< std::endl; 
            return false;
        }
    }
    return true;
}

float gpuCalculateAverageAbsoluteError(const std::vector<float>& correct,const std::vector<float>& obtained)
{
    if (correct.size() != obtained.size())
    {
        std::cout << "Correct and obtained vectors don't have same size!" << std::endl;
        return false;
    }

    float accAbsErr = 0.0f;

    for(size_t i = 0;i < correct.size();i++)
    {
        accAbsErr += std::abs(correct[i] - obtained[i]);
    }
    return accAbsErr/static_cast<float>(correct.size());
}

void printResultGpu(const std::string& message,const std::vector<float>& correct,const std::vector<float>& obtained,bool correctness,float kernelTime,float totalTime){

    const std::string upperSepSx  = "------------------ ";
    const std::string upperSepDx  = " ------------------";
    const std::string downerSep(upperSepSx.length()*2+message.length(),'-'); 

    std::cout << upperSepSx << message << upperSepDx                                       << std::endl
              << "| correct        => " << ((correctness)? "true":"false")                    << std::endl
              << "| kernel time    => " << kernelTime << " ms"                             << std::endl
              << "| total  time    => " << totalTime  << " ms"                             << std::endl
              << "| avg abs err    => " << gpuCalculateAverageAbsoluteError(correct,obtained) << std::endl
              << downerSep                                                                 << std::endl;

}
/* ========================================================================================== */


struct LUTBatchDevice{
    float* wmrSFP;
    float* wmhSFP;
    float* isoSFP;
};

#define CUDAERRCHECK(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

__global__ void commitMatrixMultiplication(
    unsigned nR,unsigned ndirs, LUTBatchDevice lutBatchesDevice,
    float* yDevice
)
{
    /* SHARED MEMORY BUFFERS */
    extern __shared__ float LUTBuffer[];

    /* IC SECTION */
    for(unsigned radii = 0 ; radii < nR ; radii++)
    {
        for(unsigned direction = 0 ; direction < ndirs ; direction++)
        {
            LUTBuffer[(radii*ndirs*SAMPLE_TILE_LENGTH) + (direction*SAMPLE_TILE_LENGTH + threadIdx.x)] = lutBatchesDevice.wmrSFP[(radii*ndirs*SAMPLE_TILE_LENGTH) + (direction*SAMPLE_TILE_LENGTH + threadIdx.x)];
        }
    }
    __syncthreads();
}

hipDeviceProp_t getCudaDeviceProps(int deviceId){

  hipDeviceProp_t deviceProps;

  hipError_t cu_err = hipGetDeviceProperties(&deviceProps, deviceId);
  if(hipSuccess != cu_err){
    printf("Unable to get hipGetDeviceProperties for device ID %d : error num %d - %s\n", deviceId, (int) cu_err, hipGetErrorString(cu_err));
    exit(EXIT_FAILURE);
  }

  return deviceProps;
}

void CommitOriginalDataStructure::gpuMatrixMultiplication()
{

    /* CHECK IF SHARED MEMORY PER BLOCK IS ENOUGH TO FIT LUT IN IT (ASSUMING THE MACHINE IS EQUIPPED WITH ONLY ONE GPU)*/
    hipDeviceProp_t deviceProps;
    CUDAERRCHECK(hipGetDeviceProperties(&deviceProps, 0))
    assert(_nR*_ndirs*SAMPLE_TILE_LENGTH*sizeof(float) < deviceProps.sharedMemPerBlock);

    hipEvent_t totalStart,totalStop;
    hipEventCreate(&totalStart);
    hipEventCreate(&totalStop);

    hipEvent_t kernelStart,kernelStop;
    hipEventCreate(&kernelStart);
    hipEventCreate(&kernelStop);

    hipEventRecord(totalStart);

    /* LUTS BATCHES ALLOCATION & COPY TO GLOBAL MEMORY */
    LUTBatchDevice* lutBatchesDevice;
    CUDAERRCHECK(hipMallocManaged((void **)&lutBatchesDevice, sizeof(LUTBatchDevice)*batchedLUTs.size()))

    for(int sampleTile = 0 ; sampleTile < batchedLUTs.size() ; sampleTile++)
    {
        float* wmrSFPDevice;
        CUDAERRCHECK(hipMalloc(&wmrSFPDevice,sizeof(float)*batchedLUTs[sampleTile].wmrSFP.size()))
        CUDAERRCHECK(hipMemcpy(wmrSFPDevice,batchedLUTs[sampleTile].wmrSFP.data(),sizeof(float)*batchedLUTs[sampleTile].wmrSFP.size(),hipMemcpyHostToDevice))

        float* wmhSFPDevice;
        CUDAERRCHECK(hipMalloc(&wmhSFPDevice,sizeof(float)*batchedLUTs[sampleTile].wmhSFP.size()))
        CUDAERRCHECK(hipMemcpy(wmhSFPDevice,batchedLUTs[sampleTile].wmhSFP.data(),sizeof(float)*batchedLUTs[sampleTile].wmhSFP.size(),hipMemcpyHostToDevice))

        float* isoSFPDevice;
        CUDAERRCHECK(hipMalloc(&isoSFPDevice,sizeof(float)*batchedLUTs[sampleTile].isoSFP.size()))
        CUDAERRCHECK(hipMemcpy(isoSFPDevice,batchedLUTs[sampleTile].isoSFP.data(),sizeof(float)*batchedLUTs[sampleTile].isoSFP.size(),hipMemcpyHostToDevice))

        lutBatchesDevice[sampleTile].wmrSFP = wmrSFPDevice;
        lutBatchesDevice[sampleTile].wmhSFP = wmhSFPDevice; 
        lutBatchesDevice[sampleTile].isoSFP = isoSFPDevice; 
    }


    /* RESULT */
    float* yDevice;
    CUDAERRCHECK(hipMalloc(&yDevice,sizeof(float)*output.size()))
    CUDAERRCHECK(hipMemset(yDevice,0.0f,sizeof(float)*output.size()))

    /* BLOCKS AND THREAD ORGANIZATION */
    const int blocks = _nV;
    const int threadsPerBlock = SAMPLE_TILE_LENGTH;

    dim3 dimGrid(blocks,1,1);
    dim3 dimBlock(threadsPerBlock,1,1);

    hipEventRecord(kernelStart);

    for(unsigned sampleTile = 0 ; sampleTile < batchedLUTs.size() ; sampleTile++)
    {
        commitMatrixMultiplication<<<dimGrid,dimBlock,_nR*_ndirs*SAMPLE_TILE_LENGTH*sizeof(float)>>>(
            _nR,_ndirs,lutBatchesDevice[sampleTile],
            yDevice
        );
    }

    hipEventRecord(kernelStop);

    /* COPYING BACK THE RESULT */
    std::vector<float> obtainedResult(output.size(),0.0f);
    CUDAERRCHECK(hipMemcpy(obtainedResult.data(),yDevice,sizeof(float)*output.size(),hipMemcpyDeviceToHost))

    /* FREEING MEMORY */
    for(int sampleTile = 0 ; sampleTile < batchedLUTs.size() ; sampleTile++)
    {
        CUDAERRCHECK(hipFree(lutBatchesDevice[sampleTile].wmrSFP))
        CUDAERRCHECK(hipFree(lutBatchesDevice[sampleTile].wmhSFP))
        CUDAERRCHECK(hipFree(lutBatchesDevice[sampleTile].isoSFP))
    }
    CUDAERRCHECK(hipFree(lutBatchesDevice))
    CUDAERRCHECK(hipFree(yDevice))

    hipEventRecord(totalStop);

    /* VERIFYING CORRECTNESS OF THE RESULT */
    hipEventSynchronize(kernelStop);
    float kernelMilliseconds = 0;
    hipEventElapsedTime(&kernelMilliseconds,kernelStart,kernelStop);
    
    hipEventSynchronize(totalStop);
    float totalMilliseconds = 0;
    hipEventElapsedTime(&totalMilliseconds,totalStart,totalStop);
    
    printResultGpu("Gpu matrix multiplication",output,obtainedResult,verifyCorrectnessGpu<float>(output,obtainedResult),kernelMilliseconds,totalMilliseconds);

    hipDeviceReset();
}