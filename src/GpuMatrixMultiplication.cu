#include "hip/hip_runtime.h"
#include "CommitOriginalDataStructure.hpp"

#include <iomanip>

/* ========================================================================================== */
template<typename T>
bool areNearlyEqualGpu(T a, T b) {
    const T normal_min = std::numeric_limits<T>::min();
    const T relative_error = 0.00001;
    if (!std::isfinite(a) || !std::isfinite(b))
    {
        return false;
    }

    T diff = std::abs(a - b);
    if (diff <= normal_min) 
        return true;

    T abs_a = std::abs(a);
    T abs_b = std::abs(b);

    return (diff / std::max(abs_a, abs_b)) <= relative_error;
}

template<typename T>
bool verifyCorrectnessGpu(const std::vector<T>& correct,const std::vector<T>& obtained)
{
    if (correct.size() != obtained.size())
    {
        std::cout << "Correct and obtained vectors don't have same size!" << std::endl;
        return false;
    }

    for(size_t i = 0;i < correct.size();i++)
    {
        if(!areNearlyEqualGpu<T>(correct[i],obtained[i]))
        {
            std::cout << std::fixed << std::setprecision(6) <<
                      "Error found : correct[" << i << "] = " << correct[i] << ", obtained[" << i << "] = " << obtained[i] << ", abs err : " << std::abs(correct[i] - obtained[i])<< std::endl; 
            return false;
        }
    }
    return true;
}

float gpuCalculateAverageAbsoluteError(const std::vector<float>& correct,const std::vector<float>& obtained)
{
    if (correct.size() != obtained.size())
    {
        std::cout << "Correct and obtained vectors don't have same size!" << std::endl;
        return false;
    }

    float accAbsErr = 0.0f;

    for(size_t i = 0;i < correct.size();i++)
    {
        accAbsErr += std::abs(correct[i] - obtained[i]);
    }
    return accAbsErr/static_cast<float>(correct.size());
}

void printResultGpu(const std::string& message,const std::vector<float>& correct,const std::vector<float>& obtained,bool correctness,float kernelTime,float totalTime){

    const std::string upperSepSx  = "------------------ ";
    const std::string upperSepDx  = " ------------------";
    const std::string downerSep(upperSepSx.length()*2+message.length(),'-'); 

    std::cout << upperSepSx << message << upperSepDx                                       << std::endl
              << "| correct        => " << ((correctness)? "true":"false")                    << std::endl
              << "| kernel time    => " << kernelTime << " ms"                             << std::endl
              << "| total  time    => " << totalTime  << " ms"                             << std::endl
              << "| avg abs err    => " << gpuCalculateAverageAbsoluteError(correct,obtained) << std::endl
              << downerSep                                                                 << std::endl;

}
/* ========================================================================================== */

#define CUDAERRCHECK(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

/*
    QUADDRO P6000 => 
                        max_threads_per_sm : 2048
                        max_blocks_per_sm  : 32
                        -------------------------
                        threads per block to achieve max occupancy : 64

                        n_sm : 30
                        -------------------------
                        to achieve full GPU occupancy : 960 (or a multiple of it) blocks of 64 threads

    RTX 2060      => 
                        max_threads_per_sm : 1024
                        max_blocks_per_sm  : 16
                        -------------------------
                        threads per block to achieve max occupancy : 64

                        n_sm : 30
                        -------------------------
                        to achieve full GPU occupancy : 480 (or a multiple of it) blocks of 64 threads 
*/

__global__ void commitMatrixMultiplication(
    int nS,
    int nV,
    uint32_t* icfDevice, uint32_t* icvDevice, uint16_t* icoDevice, float* iclDevice, int nR,int nF,
    uint32_t* ecvDevice, uint16_t* ecoDevice, int nT,int nE,
    uint32_t* isovDevice, int nI,
    float* wmrSFPDevice, float* wmhSFPDevice, float* isoSFPDevice,int ndirs,
    int* icIndexesDevice, int* ecIndexesDevice,
    float* xDevice,
    float* yDevice
)
{
    const int voxel  = blockIdx.x;
    const int sample = threadIdx.x;

    /* IC SEGMENTS TO ELABORATE */
    const int startIcSegment = (voxel==0)?0:icIndexesDevice[voxel-1];
    const int endIcSegment   = icIndexesDevice[voxel];

    /* EC SEGMENTS TO ELABORATE */
    const int startEcSegment = (voxel==0)?0:ecIndexesDevice[voxel-1];
    const int endEcSegment   = ecIndexesDevice[voxel];

    float accumulator = 0.0f;

    /* IC */
    for (int radii = 0; radii < nR; radii++)
    {
        int lookupTableOffset = radii*ndirs*nS;

        for(int icsegment = startIcSegment; icsegment < endIcSegment; icsegment++)
        {
            accumulator += xDevice[icfDevice[icsegment] + radii]*wmrSFPDevice[lookupTableOffset + icoDevice[icsegment] * nS + sample]*iclDevice[icsegment];
        }
    }
    /* EC */
    for (int tortuosity = 0; tortuosity < nT; tortuosity++)
    {
        int lookupTableOffset = tortuosity*ndirs*nS;
        int xIndex = nR*nF + tortuosity*nE + startEcSegment;

        for(int ecsegment = startEcSegment; ecsegment < endEcSegment; ecsegment++)
        {
            accumulator += xDevice[xIndex]*wmhSFPDevice[lookupTableOffset + ecoDevice[ecsegment] * nS + sample];
            xIndex++;
        }
    }
    /* ISO */
    for (int iso = 0; iso < nI; iso++)
    {
        accumulator += xDevice[(nR*nF + nT*nE + voxel) + iso*nV]*isoSFPDevice[iso * nS + sample];
    }
    yDevice[voxel * nS + sample] = accumulator;
}

void CommitOriginalDataStructure::gpuMatrixMultiplication()
{
    hipEvent_t totalStart,totalStop;
    hipEventCreate(&totalStart);
    hipEventCreate(&totalStop);

    hipEvent_t kernelStart,kernelStop;
    hipEventCreate(&kernelStart);
    hipEventCreate(&kernelStop);

    hipEventRecord(totalStart);

    /* IC */
    uint32_t* icfDevice; uint32_t* icvDevice; uint16_t* icoDevice; float* iclDevice;

    CUDAERRCHECK(hipMalloc(&icfDevice,sizeof(uint32_t)*icf.size()))
    CUDAERRCHECK(hipMalloc(&icvDevice,sizeof(uint32_t)*icv.size()))
    CUDAERRCHECK(hipMalloc(&icoDevice,sizeof(uint16_t)*ico.size()))
    CUDAERRCHECK(hipMalloc(&iclDevice,sizeof(float)*icl.size()))

    CUDAERRCHECK(hipMemcpy(icfDevice,icf.data(),sizeof(uint32_t)*icf.size(),hipMemcpyHostToDevice))
    CUDAERRCHECK(hipMemcpy(icvDevice,icv.data(),sizeof(uint32_t)*icv.size(),hipMemcpyHostToDevice))
    CUDAERRCHECK(hipMemcpy(icoDevice,ico.data(),sizeof(uint16_t)*ico.size(),hipMemcpyHostToDevice))
    CUDAERRCHECK(hipMemcpy(iclDevice,icl.data(),sizeof(float)*icl.size(),hipMemcpyHostToDevice))

    /* EC */
    uint32_t* ecvDevice; uint16_t* ecoDevice;

    CUDAERRCHECK(hipMalloc(&ecvDevice,sizeof(uint32_t)*ecv.size()))
    CUDAERRCHECK(hipMalloc(&ecoDevice,sizeof(uint16_t)*eco.size()))

    CUDAERRCHECK(hipMemcpy(ecvDevice,ecv.data(),sizeof(uint32_t)*ecv.size(),hipMemcpyHostToDevice))
    CUDAERRCHECK(hipMemcpy(ecoDevice,eco.data(),sizeof(uint16_t)*eco.size(),hipMemcpyHostToDevice))

    /* ISO */
    uint32_t* isovDevice;

    CUDAERRCHECK(hipMalloc(&isovDevice,sizeof(uint32_t)*isov.size()))

    CUDAERRCHECK(hipMemcpy(isovDevice,isov.data(),sizeof(uint32_t)*isov.size(),hipMemcpyHostToDevice))

    /* LOOKUP TABLE */
    float* wmrSFPDevice;float* wmhSFPDevice;float* isoSFPDevice;

    CUDAERRCHECK(hipMalloc(&wmrSFPDevice,sizeof(float)*wmrSFP.size()))
    CUDAERRCHECK(hipMalloc(&wmhSFPDevice,sizeof(float)*wmhSFP.size()))
    CUDAERRCHECK(hipMalloc(&isoSFPDevice,sizeof(float)*isoSFP.size()))

    CUDAERRCHECK(hipMemcpy(wmrSFPDevice,wmrSFP.data(),sizeof(float)*wmrSFP.size(),hipMemcpyHostToDevice))
    CUDAERRCHECK(hipMemcpy(wmhSFPDevice,wmhSFP.data(),sizeof(float)*wmhSFP.size(),hipMemcpyHostToDevice))
    CUDAERRCHECK(hipMemcpy(isoSFPDevice,isoSFP.data(),sizeof(float)*isoSFP.size(),hipMemcpyHostToDevice))

    /* HELPER ARRAYS */
    int* icIndexesDevice;int* ecIndexesDevice;

    CUDAERRCHECK(hipMalloc(&icIndexesDevice,sizeof(int)*icIndexes.size()))
    CUDAERRCHECK(hipMalloc(&ecIndexesDevice,sizeof(int)*ecIndexes.size()))

    CUDAERRCHECK(hipMemcpy(icIndexesDevice,icIndexes.data(),sizeof(int)*icIndexes.size(),hipMemcpyHostToDevice))
    CUDAERRCHECK(hipMemcpy(ecIndexesDevice,ecIndexes.data(),sizeof(int)*ecIndexes.size(),hipMemcpyHostToDevice))

    /* INPUT */
    float* xDevice;
    
    CUDAERRCHECK(hipMalloc(&xDevice,sizeof(float)*input.size()))

    CUDAERRCHECK(hipMemcpy(xDevice,input.data(),sizeof(float)*input.size(),hipMemcpyHostToDevice))

    /* RESULT */
    float* yDevice;

    CUDAERRCHECK(hipMalloc(&yDevice,sizeof(float)*output.size()))
    
    CUDAERRCHECK(hipMemset(yDevice,0.0f,sizeof(float)*output.size()))

    /* BLOCKS AND THREAD ORGANIZATION */
    const int blocks = _nV;
    const int threadsPerBlock = _nS;

    dim3 dimGrid(blocks,1,1);
    dim3 dimBlock(threadsPerBlock,1,1);

    hipEventRecord(kernelStart);
    commitMatrixMultiplication<<<dimGrid,dimBlock>>>(
        _nS,
        _nV,
        icfDevice,icvDevice,icoDevice,iclDevice,_nR,_nF,
        ecvDevice,ecoDevice,_nT,_nE,
        isovDevice,_nI,
        wmrSFPDevice,wmhSFPDevice,isoSFPDevice,_ndirs,
        icIndexesDevice,ecIndexesDevice,
        xDevice,
        yDevice
    );
    hipEventRecord(kernelStop);

    /* COPYING BACK THE RESULT */
    std::vector<float> obtainedResult(output.size(),0.0f);
    CUDAERRCHECK(hipMemcpy(obtainedResult.data(),yDevice,sizeof(float)*output.size(),hipMemcpyDeviceToHost))

    /* FREEING MEMORY */
    hipFree(icfDevice);hipFree(icvDevice);hipFree(icoDevice);hipFree(iclDevice);
    hipFree(ecvDevice);hipFree(ecoDevice);
    hipFree(isovDevice);
    hipFree(wmrSFPDevice);hipFree(wmhSFPDevice);hipFree(isoSFPDevice);
    hipFree(icIndexesDevice);hipFree(ecIndexesDevice);

    hipEventRecord(totalStop);

    /* VERIFYING CORRECTNESS OF THE RESULT */
    hipEventSynchronize(kernelStop);
    float kernelMilliseconds = 0;
    hipEventElapsedTime(&kernelMilliseconds,kernelStart,kernelStop);
    
    hipEventSynchronize(totalStop);
    float totalMilliseconds = 0;
    hipEventElapsedTime(&totalMilliseconds,totalStart,totalStop);
    
    printResultGpu("Gpu matrix multiplication",output,obtainedResult,verifyCorrectnessGpu<float>(output,obtainedResult),kernelMilliseconds,totalMilliseconds);

    hipDeviceReset();
}